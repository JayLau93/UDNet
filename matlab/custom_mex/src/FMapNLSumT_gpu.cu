#include "hip/hip_runtime.h"
/* // Compilation instructions
 * mex -v FMapNLSumT_gpu.cu -largeArrayDims
 * CFLAGS="\$CFLAGS -std=c++11 -g" LDFLAGS='\$LDFLAGS -Wl,-rpath,/usr/local/cuda/lib'
 * -I/usr/local/include/
 * -L/usr/local/cuda/lib -lcudart -lstdc++ -lc */


/* Let us assume that X is of size (H * W) x D x N, idx is of size 
 * (H * W) x Nb x N (computed by misc.patchMatch) where 
 * the elements of idx have values in the range [1, H*W] and Weights is of 
 * size (H x W) x (D x Nb) x N or (H x W) x Nb x N or 
 * D x Nb or Nb. In the second case the same weights are applied on the 
 * D feature maps at each spatial dimension, while in the last two cases  
 * the same weights are applied to the H x W spatial elements.
 *
 * Then the output Y will be of size (H * W) x D x N where 
 *
 *            Nb
 * Y(i,d,n) = Sum W(i,idx_w(d,r),n)*X(idx(i,r,n),d,n), i=1:H*W, d=1:D, n=1:N
 *            r=1
 * and idx_w(d,r) = r*D + d
 *
 *          Nb
 * Y(ind_y) = Sum W(ind_w)*X(ind_x) where
 *          r=1
 * ind_y = i + d*H*W + n*H*W*D, 
 * ind_w = i + (d+r*D)*H*W + n*H*W*D*Nb,  (W is of size HW x D x Nb x N)
 * ind_x = idx[ind_r] + d*H*W + n*H*W*D where 
 * ind_r = i + r*H*W + n*H*W*Nb
 *
 * To check the correctness of the computation do the following in Matlab:
 * 
 * H=10;W=20;D=3;N=2;Nb=4;
 * X=randn(H,W,D,N); idx=randsrc(H*W*Nb*N,1,[1:H*W]); idx = reshape(idx,H,W,Nb,N);
 * idx(:,:,1,1) = reshape(1:H*W,H,W); idx(:,:,1,2)=idx(:,:,1,1); idx = uint32(idx);
 * Weights = randn(H,W,Nb,N);
 * [X,idx,Weights] = misc.move_data('gpu',X,idx,Weights);
 *
 *
 * Y = zeros(size(X),'like',X);
 * for i = 1:H, for j=1:W, for d=1:D, for n=1:N, for r=1:Nb, 
 * Y(i,j,d,n) = Y(i,j,d,n) + Weights(i,j,r,n)*X(idx(i,j,r,n)+(d-1)*H*W+(n-1)*H*W*D);
 * end,end,end,end,end
 *
 * Y2 = FMapNLSum_gpu(X,Weights,idx-1); % We substract 1 from idx since
 * % in C++ the starting element is zero and not 1.
 *
 * e = Y-Y2; max(e(:)), min(e(:))
 *
 * 
 * Now in order to compute the adjoint operation let us consider the following
 * example where H*W = 6, D=3, Nb = 2 and N = 1
 *
 * Let X = [ x1:D    idx = [1 3 4   Weights = [ W1:D,1 W1:D,2 W1:D,3
 *           x2:D           2 1 6               W2:D,1 W2:D,2 W2:D,3
 *           x3:D           3 5 2               W3:D,1 W3:D,2 W3:D,3
 *           x4:D           4 1 6               W4:D,1 W4:D,2 W4:D,3
 *           x5:D           5 2 3               W5:D,1 W5:D,2 W5:D,3
 *           x6:D ]         6 4 3]              W6:D,1 W6:D,2 W6:D,3]
 *
 *  xk:D ==> It denotes all the D channels for the k spatial coordinate
 *  W(k,r):D ==> It denotes the D-dimensional weight vector for the rth 
 *  closest neighbor.
 *        
 *  Based on the above, the forward operation is given by:
 *
 *  Y = [ x1:D .* W1:D,1 + x3:D .* W1:D,2 + x4:D .* W1:D,3
 *        x2:D .* W2:D,1 + x1:D .* W2:D,2 + x6:D .* W2:D,3
 *        x3:D .* W3:D,1 + x5:D .* W3:D,2 + x2:D .* W3:D,3
 *        x4:D .* W4:D,1 + x1:D .* W4:D,2 + x6:D .* W4:D,3
 *        x5:D .* W5:D,1 + x2:D .* W5:D,2 + x3:D .* W5:D,3
 *        x6:D .* W6:D,1 + x4:D .* W6:D,2 + x3:D .* W6:D,3 ]
 *
 *  and the adjoint is given by
 *
 *  Z = [ y1:D .* W1:D,1 + y2:D .* W2:D,2 + y4:D .* W4:D,2
 *        y2:D .* W2:D,1 + y3:D .* W3:D,3 + y5:D .* W5:D,2
 *        y1:D .* W1:D,2 + y3:D .* W3:D,1 + y5:D .* W5:D,3 + y6:D .* W6:D,3
 *        y1:D .* W1:D,3 + y4:D .* W4:D,1 + y6:D .* W6:D,2
 *        y2:D .* W2:D,2 + y5:D .* W5:D,1
 *        y2:D .* W2:D,3 + y4:D .* W4:D,3 + y6:D .* W6:D,1 ]
 *
 *  We observe that zk:D,n is given as the weighted sum of Wl:D,m,n and yl:D,n 
 *  where l,m,n are the row, column and slice where the number k+N*n is 
 *  found in idx. (In this example we considered N=1 but this holds true
 *  for N >= 1). 
 *  
 *  NOTE!!!!
 *  In order the adjoint to correctly work for multiple images 
 *  we must transform the idx computed by misc.patchMatch so that 
 *  idx_new(:,:,n) = idx(:,:,n) + (n-1)*H*W. This is taken care by the 
 *  surrogate function FMapNLSumT_helper.m
 *
 *  [widx,n,I]=misc.FMapNLSumT_helper(idx);
 *  Z = FMapNLSumT_gpu(Y,Weights,widx-1,n,I-1);
 *  e = Z(:)'* X(:) - Y(:)'*Y(:) // This should be close to zero.
 * /

/* In a mxArray to access the element X[i][j][z] you can do it by referring
   to the element X[i+j*dims[0]+z*dims[0]*dims[1]] */


#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#if __CUDA_ARCH__ >= 200
#define VL_CUDA_NUM_THREADS 1024
#else
#define VL_CUDA_NUM_THREADS 512
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line)
{
  if (code != hipSuccess)
  {
    char *err_str = new char[1000];
    sprintf(err_str,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    mexErrMsgTxt(err_str);
    delete [] err_str;
  }
}

inline unsigned int divideUpwards(unsigned int a, unsigned int b)
{
  return (a + b - 1) / b ;
}

__device__ void index_map(const mwSize *c, const mwSize index, mwSize *ind){
  
  // From ndx = i + j*H + d*H*W + n*H*W*D and c = [H*W, H*W*D, Nb]
  // we want to recover ind[0] = i + j*H, ind[1]=d and ind[2]=n
  
  ind[0] = index % c[0]; // i+j*H
  ind[1] = ((index - ind[0]) % c[1]) / c[0]; // d
  ind[2] = index / c[1]; // n
  
}


template <typename T>
        __global__ void FMapNLSumT_gpu_kernel(
        const T *X, const T *W, T* Y,
        const unsigned int *n_table,
        const unsigned int *I,
        const unsigned int *idx,
        const mwSize *c,
        const bool FMapSharedWeights,
        const bool SpatialSharedWeights,
        size_t numElements)
{
  
  size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  
  if (index < numElements){
    mwSize ind[3];
    mwSize coord, coord_X, coord_W;
    index_map(c,index,ind);
    // For any channel k of X(i,j,k,n) we have to use n_table(i,j,n) and
    // I(i,j,n).
    // index_n = i+j*H+n*H*W
    mwSize index_n = ind[0]+c[0]*ind[2];
  
    for (mwSize k = 0; k < n_table[index_n]; ++k){
    
      coord = idx[I[index_n]+k]; // coord = i + j*H + r*H*W + n*H*W*Nb
      coord_X = (coord % c[0])+c[0]*ind[1]+c[1]*ind[2]; // coord_X = i + j*H + d*H*W + n*H*W*N
    
      if (!FMapSharedWeights && !SpatialSharedWeights){
        // coord_W  = i + j*H + d*H*W + r*H*W*D + n*H*W*D*Nb
        coord_W = (coord % c[0]) + c[0]*ind[1] + c[1]*((coord / c[0]) % c[2]) + ind[2]*c[1]*c[2];
      }
      else if (FMapSharedWeights && !SpatialSharedWeights){
        // coord_W = i + j*H + r*H*W + n*H*W*Nb
        coord_W = coord;
      }
      else if (!FMapSharedWeights && SpatialSharedWeights){
        // coord_W = d + D*r
        coord_W = ind[1]+(c[1]/c[0])*((coord / c[0]) % c[2]);
      }
      else{
        // coord_W = r
        coord_W = (coord / c[0]) % c[2];
      }
    
      Y[index] += X[coord_X]*W[coord_W];
    }       
  }
}


template <typename T> 
static inline hipError_t FMapNLSumT_gpu(
        const T *X, const T *W, T *Y, 
        const unsigned int *n_table,
        const unsigned int *I,
        const unsigned int *idx,
        const mwSize *c,
        const bool FMapSharedWeights,
        const bool SpatialSharedWeights,        
        size_t numElements)
{  
 FMapNLSumT_gpu_kernel<T>
 <<< divideUpwards(numElements, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
 (X,W,Y,n_table,I,idx,c,FMapSharedWeights,SpatialSharedWeights,numElements);
 return hipPeekAtLastError(); 
}


void mexFunction(int nlhs, mxArray *plhs[],
        int nrhs, mxArray const *prhs[])
{
  
  // Create Error Messages.
  char const * const errId = "FMapNLSumT_gpu:InvalidInput";
  char const * const errMsg_narg = "Invalid input: Five input "
          "arguments are expected.";
  char const * const errMsg_ty = "Invalid input: Input has unsupported type.";
  char const * const errMsg_args = "Invalid input: All the inputs must be between a 1-D to 4-D array.";
  char const * const errMsg_dims = "Invalid input: Dimensions mismatch.";
  char const * const errMsg_type = "Invalid input: Inputs 1, 2 must be of the same data type.";
  char const * const errMsg_type2 = "Invalid input: Inputs 3-5 must be of type 'uint32'.";
  
  
  /* Initialize the MathWorks GPU API. */
  mxInitGPU(); 
  
  if (nrhs != 5)
    mexErrMsgIdAndTxt(errId, errMsg_narg);
  
  // Get the inputs
  // X 4D input [H W D N] where D is the number of channels and N the
  // number of images.
  const mxGPUArray *X_mx = mxGPUCreateFromMxArray(prhs[0]);
  // 4D array [H W (D*Nb | Nb) N] or 1D array [D*Nb | Nb]
  const mxGPUArray *Weights_mx  = mxGPUCreateFromMxArray(prhs[1]);
  // 1D array [H*W*Nb*N]
  const mxGPUArray *idx_mx  = mxGPUCreateFromMxArray(prhs[2]);
  // 1D array [H*W*N]
  const mxGPUArray *n_mx  = mxGPUCreateFromMxArray(prhs[3]);
  // 1D array [H*W*N]
  const mxGPUArray *I_mx  = mxGPUCreateFromMxArray(prhs[4]);
  
  const mxClassID cid = mxGPUGetClassID(X_mx);
  if (cid != mxGPUGetClassID(Weights_mx))
    mexErrMsgIdAndTxt(errId,errMsg_type);
  
  const mxClassID cid2 = mxGPUGetClassID(idx_mx);
  if (cid2 != mxUINT32_CLASS)
    mexErrMsgIdAndTxt(errId,errMsg_type2);
  
  if (cid2 != mxGPUGetClassID(n_mx) || cid2 != mxGPUGetClassID(I_mx))
    mexErrMsgIdAndTxt(errId,errMsg_type2);
  
  // X 4D Array [H W D N]
  const mwSize X_ndims = mxGPUGetNumberOfDimensions(X_mx);
  if ( X_ndims < 1 || X_ndims > 4)
    mexErrMsgIdAndTxt(errId, errMsg_args);
  
  const mwSize *X_dims = mxGPUGetDimensions(X_mx);
  const mwSize H = X_dims[0];
  const mwSize W = (X_ndims < 2) ? 1 : X_dims[1];
  const mwSize D = (X_ndims < 3) ? 1 : X_dims[2];
  const mwSize N = (X_ndims < 4) ? 1 : X_dims[3];
  
  mwSize numElements=H*W*D*N;
  
  // idx 1D Array [H*W*Nb*N]
  mwSize idxElements = mxGPUGetNumberOfElements(idx_mx);
  const mwSize Nb = idxElements/(H*W*N);
  if (idxElements % (H*W*N) != 0)
    mexErrMsgIdAndTxt(errId,errMsg_dims);
  const unsigned int *idx_ptr = static_cast<const unsigned int*>(mxGPUGetDataReadOnly(idx_mx));
  
  // Weights : 4D Array [H W D*Nb N] or [H W Nb N] or 2D Array [D*Nb] or [Nb]
  const mwSize *Weights_dims = mxGPUGetDimensions(Weights_mx);
  const mwSize W_ndims = mxGPUGetNumberOfDimensions(Weights_mx);
  if ( W_ndims < 1 || W_ndims > 4)
    mexErrMsgIdAndTxt(errId, errMsg_args);
  
  const mwSize numWeightElements = mxGPUGetNumberOfElements(Weights_mx);
    
  bool SpatialSharedWeights = false; 
  bool FMapSharedWeights = false;  
  if (W_ndims == 2){
    SpatialSharedWeights = true; // Weights 1D Array of D*Nb or Nb elements
    
    if (numWeightElements == Nb)
      FMapSharedWeights = true;
    
    if (!FMapSharedWeights && numWeightElements != Nb*D)
      mexErrMsgIdAndTxt(errId, errMsg_dims);    
  }
  
  if (W_ndims > 2){
    if ((Weights_dims[0] != H) || (Weights_dims[1] != W))
      mexErrMsgIdAndTxt(errId, errMsg_dims);
    
    if (Weights_dims[2] == Nb)
      FMapSharedWeights = true;
    
    if (!FMapSharedWeights && Weights_dims[2] != Nb*D)
      mexErrMsgIdAndTxt(errId, errMsg_dims);
    
    if (W_ndims < 4){
      if (N != 1)
        mexErrMsgIdAndTxt(errId, errMsg_dims);
    }
    else{
      if (Weights_dims[3] != N)
        mexErrMsgIdAndTxt(errId, errMsg_dims);
    }
  }  
  
  // n 1D Array [H*W*N]
  if (mxGPUGetNumberOfElements(n_mx)!=H*W*N)
    mexErrMsgIdAndTxt(errId,errMsg_dims);
  const unsigned int *n_ptr = static_cast<const unsigned int*>(mxGPUGetDataReadOnly(n_mx));
  
  // I 1D Array [H*W*N]
  if (mxGPUGetNumberOfElements(I_mx)!=H*W*N)
    mexErrMsgIdAndTxt(errId,errMsg_dims);
  const unsigned int *I_ptr = static_cast<const unsigned int*>(mxGPUGetDataReadOnly(I_mx));
  
  
  // Create output
  
  // Y [H, W, D, N]
  const mwSize Y_dims[]={H, W, D, N};
  mxGPUArray *Y_mx = mxGPUCreateGPUArray(4, Y_dims, cid, mxREAL, MX_GPU_INITIALIZE_VALUES);
  
  const mwSize c_ptr[3]={H*W,H*W*D,Nb};
  mwSize *d_c;
  hipMalloc(&d_c,3*sizeof(mwSize));
  hipMemcpy(d_c,c_ptr,3*sizeof(mwSize),hipMemcpyHostToDevice);
  
  if (cid == mxDOUBLE_CLASS){
    const double *X_ptr = static_cast<const double*>(mxGPUGetDataReadOnly(X_mx));
    const double *Weights_ptr = static_cast<const double*>(mxGPUGetDataReadOnly(Weights_mx));
    double *Y_ptr = static_cast<double*>(mxGPUGetData(Y_mx));
    
    gpuErrchk(FMapNLSumT_gpu<double>(X_ptr,Weights_ptr,Y_ptr,n_ptr,
    I_ptr,idx_ptr,d_c,FMapSharedWeights,SpatialSharedWeights,numElements));
  }
  else if (cid == mxSINGLE_CLASS){
    const float *X_ptr = static_cast<const float*>(mxGPUGetDataReadOnly(X_mx));
    const float *Weights_ptr = static_cast<const float*>(mxGPUGetDataReadOnly(Weights_mx));
    float *Y_ptr = static_cast<float*>(mxGPUGetData(Y_mx));
    
    gpuErrchk(FMapNLSumT_gpu<float>(X_ptr,Weights_ptr,Y_ptr,n_ptr,
    I_ptr,idx_ptr,d_c,FMapSharedWeights,SpatialSharedWeights,numElements));    
  }
  else
    mexErrMsgIdAndTxt(errId, errMsg_ty);
  
  
  plhs[0] = mxGPUCreateMxArrayOnGPU(Y_mx);
  
  mxGPUDestroyGPUArray(X_mx);
  mxGPUDestroyGPUArray(Weights_mx);
  mxGPUDestroyGPUArray(Y_mx);
  mxGPUDestroyGPUArray(idx_mx);
  mxGPUDestroyGPUArray(n_mx);
  mxGPUDestroyGPUArray(I_mx);  
}










