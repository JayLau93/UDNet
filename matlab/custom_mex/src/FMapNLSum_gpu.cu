#include "hip/hip_runtime.h"
/* // Compilation instructions
 * mex -v FMapNLSum_gpu.cu -largeArrayDims
 * CFLAGS="\$CFLAGS -std=c++11 -g" LDFLAGS='\$LDFLAGS -Wl,-rpath,/usr/local/cuda/lib'
 * -I/usr/local/include/ 
 * -L/usr/local/cuda/lib -lcudart -lstdc++ -lc */

 /* Let us assume that X is of size (H * W) x D x N, idx is of size 
 * (H * W) x Nb x N (computed by misc.patchMatch) where 
 * the elements of idx have values in the range [1, H*W] and Weights is of 
 * size (H x W) x (D x Nb) x N or (H x W) x Nb x N or 
 * D x Nb or Nb. In the second case the same weights are applied on the 
 * D feature maps at each spatial dimension, while in the last two cases  
 * the same weights are applied to the H x W spatial elements.
 *
 * Then the output Y will be of size (H * W) x D x N where 
 *
 *            Nb
 * Y(i,d,n) = Sum W(i,idx_w(d,r),n)*X(idx(i,r,n),d,n), i=1:H*W, d=1:D, n=1:N
 *            r=1
 * and idx_w(d,r) = r*D + d
 *
 *          Nb
 * Y(ind_y) = Sum W(ind_w)*X(ind_x) where
 *          r=1
 * ind_y = i + d*H*W + n*H*W*D, 
 * ind_w = i + (d+r*D)*H*W + n*H*W*D*Nb,  (W is of size HW x D x Nb x N)
 * ind_x = idx[ind_r] + d*H*W + n*H*W*D where 
 * ind_r = i + r*H*W + n*H*W*Nb
 *
 * To check the correctness of the computation do the following in Matlab:
 * 
 * H=10;W=20;D=3;N=2;Nb=4;
 * X=randn(H,W,D,N); idx=randsrc(H*W*Nb*N,1,[1:H*W]); idx = reshape(idx,H,W,Nb,N);
 * idx(:,:,1,1) = reshape(1:H*W,H,W); idx(:,:,1,2)=idx(:,:,1,1); idx = uint32(idx);
 * Weights = randn(H,W,Nb,N);
 *[X,idx,Weights] = misc.move_data('gpu',X,idx,Weights);
 *
 * Y = zeros(size(X),'like',X);
 * for i = 1:H, for j=1:W, for d=1:D, for n=1:N, for r=1:Nb, 
 * Y(i,j,d,n) = Y(i,j,d,n) + Weights(i,j,r,n)*X(idx(i,j,r,n)+(d-1)*H*W+(n-1)*H*W*D);
 * end,end,end,end,end
 *
 * Y2 = FMapNLSum_gpu(X,Weights,idx-1); % We substract 1 from idx since
 * % in C++ the starting element is zero and not 1.
 *
 * e = Y-Y2; max(e(:)), min(e(:))
 *
 * 
 * Now in order to compute the adjoint operation let us consider the following
 * example where H*W = 6, D=3, Nb = 2 and N = 1
 *
 * Let X = [ x1:D    idx = [1 3 4   Weights = [ W1:D,1 W1:D,2 W1:D,3
 *           x2:D           2 1 6               W2:D,1 W2:D,2 W2:D,3
 *           x3:D           3 5 2               W3:D,1 W3:D,2 W3:D,3
 *           x4:D           4 1 6               W4:D,1 W4:D,2 W4:D,3
 *           x5:D           5 2 3               W5:D,1 W5:D,2 W5:D,3
 *           x6:D ]         6 4 3]              W6:D,1 W6:D,2 W6:D,3]
 *
 *  xk:D ==> It denotes all the D channels for the k spatial coordinate
 *  W(k,r):D ==> It denotes the D-dimensional weight vector for the rth 
 *  closest neighbor.
 *        
 *  Based on the above, the forward operation is given by:
 *
 *  Y = [ x1:D .* W1:D,1 + x3:D .* W1:D,2 + x4:D .* W1:D,3
 *        x2:D .* W2:D,1 + x1:D .* W2:D,2 + x6:D .* W2:D,3
 *        x3:D .* W3:D,1 + x5:D .* W3:D,2 + x2:D .* W3:D,3
 *        x4:D .* W4:D,1 + x1:D .* W4:D,2 + x6:D .* W4:D,3
 *        x5:D .* W5:D,1 + x2:D .* W5:D,2 + x3:D .* W5:D,3
 *        x6:D .* W6:D,1 + x4:D .* W6:D,2 + x3:D .* W6:D,3 ]
 *
 *  and the adjoint is given by
 *
 *  Z = [ y1:D .* W1:D,1 + y2:D .* W2:D,2 + y4:D .* W4:D,2
 *        y2:D .* W2:D,1 + y3:D .* W3:D,3 + y5:D .* W5:D,2
 *        y1:D .* W1:D,2 + y3:D .* W3:D,1 + y5:D .* W5:D,3 + y6:D .* W6:D,3
 *        y1:D .* W1:D,3 + y4:D .* W4:D,1 + y6:D .* W6:D,2
 *        y2:D .* W2:D,2 + y5:D .* W5:D,1
 *        y2:D .* W2:D,3 + y4:D .* W4:D,3 + y6:D .* W6:D,1 ]
 *
 *  We observe that zk:D,n is given as the weighted sum of Wl,m,n and yl:D,n 
 *  where l,m,n are the row, column and slice where the number k+N*n is 
 *  found in idx. (In this example we considered N=1 but this holds true
 *  for N >= 1). 
 *  
 *  NOTE!!!!
 *  In order the adjoint to correctly work for multiple images 
 *  we must transform the idx computed by misc.patchMatch so that 
 *  idx_new(:,:,n) = idx(:,:,n) + (n-1)*H*W. This is taken care by the 
 *  surrogate function WeightedMapSumT_helper.m
 *
 *  [widx,n,I]=misc.FMapNLSumT_helper(idx);
 *  Z = FMapNLSumT_gpu(Y,Weights,widx-1,n,I-1);
 *  e = Z(:)'* X(:) - Y(:)'*Y(:) // This should be close to zero.
 * /

/* In a mxArray to access the element X[i][j][z] you can do it by referring
   to the element X[i+j*dims[0]+z*dims[0]*dims[1]]
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#if __CUDA_ARCH__ >= 200
#define VL_CUDA_NUM_THREADS 1024
#else
#define VL_CUDA_NUM_THREADS 512
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line)
{
  if (code != hipSuccess)
  {
    char *err_str = new char[1000];
    sprintf(err_str,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    mexErrMsgTxt(err_str);
    delete [] err_str;
  }
}

inline unsigned int divideUpwards(unsigned int a, unsigned int b)
{
  return (a + b - 1) / b ;
}



__device__ void index_map(const mwSize *c, const mwSize ndx, 
    const mwSize r, mwSize *ind, 
    const bool FMapSharedWeights, 
    const bool SpatialSharedWeights){
  
  // From ndx = i + j*H + d*H*W + n*H*W*D and c = [H*W, H*W*D, D, Nb]
  // we want to recover :
  // ind[0] = i + j*H + d*H*W + n*H*W*Nb and
  // ind[1] = d*H*W + n*H*W*D
  // ind[2] (The form of the last index depends on the bool variables)
  
  // if FMap = false and Spatial = false
  // ind[2] = i + j*H + d*H*W + r*H*W*D + n*H*W*D*Nb 
  
  // if FMap = true and Spatial = false 
  // ind[2] = i + j*H + r*H*W + n*H*W*Nb 
  
  // if FMap = false and Spatial = true
  // ind[2] = d + r*D
  
  // if FMap = true and Spatial = true          
  // ind[2] = r
  
  mwSize i, d, n;
  i = ndx % c[0];
  d = ((ndx % c[1]) - i) / c[0];
  n = ndx / c[1];
  
  ind[0] = i + r*c[0] + n*c[0]*c[3];  // ind[0] = i + r*H*W + n*H*W*Nb, 
  ind[1] = ndx - i; // ind[1] = d*H*W + n*H*W*D = H*W(d+n*D)
 
  if (!FMapSharedWeights && !SpatialSharedWeights)
    ind[2] = ndx % c[1] + r*c[1] + n*c[1]*c[3];
  else if (FMapSharedWeights && !SpatialSharedWeights)
    ind[2] = i + r*c[0] + n*c[0]*c[3];
  else if (!FMapSharedWeights && SpatialSharedWeights)
    ind[2] = d + r*c[2];
  else 
    ind[2] = r;           
}

template <typename T>
__global__ void FMapNLSum_gpu_kernel(
        const T *X, 
        const T *W, T* Y, 
        const unsigned int *idx, 
        const mwSize *c, 
        const mwSize Nb,
        const bool FMapSharedWeights,
        const bool SpatialSharedWeights,
        size_t numElements)
{  
  
  size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  
  if (index < numElements){
  
  mwSize ind[3];
  for (mwSize r = 0; r < Nb; ++r)
  {
    index_map(c,index,r,ind,FMapSharedWeights,SpatialSharedWeights); 
    // r=idx[ind[0]]+ind[1] = idx[i][j][r][n] + k*H*W + n*H*W*D
    Y[index] += W[ind[2]]*X[idx[ind[0]]+ind[1]];
  }
 }
  
}

template <typename T> 
static inline hipError_t FMapNLSum_gpu(
        const T *X, 
        const T *W, T *Y, 
        const unsigned int *idx,
        const mwSize *c, 
        const mwSize Nb,
        const bool FMapSharedWeights,
        const bool SpatialSharedWeights,        
        size_t numElements)
{  
 FMapNLSum_gpu_kernel<T>
 <<< divideUpwards(numElements, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
 (X,W,Y,idx,c,Nb,FMapSharedWeights,SpatialSharedWeights,numElements);
 return hipPeekAtLastError(); 
}



void mexFunction(int nlhs, mxArray *plhs[],
        int nrhs, mxArray const *prhs[])
{
  
  // Create Error Messages.
  char const * const errId = "FMapNLSum_gpu:InvalidInput";
  char const * const errMsg_narg = "Invalid input: Three input "
          "arguments are expected.";
  char const * const errMsg_ty = "Invalid input: Input has unsupported type.";
  char const * const errMsg_args = "Invalid input: All the inputs must be between a 1-D to 4-D array.";
  char const * const errMsg_dims = "Invalid input: Dimensions mismatch.";
  char const * const errMsg_type = "Invalid input: Inputs 1, 2 must be of the same data type.";
  char const * const errMsg_gpu = "Invalid input: Inputs must be gpuArrays.";
  
  
  /* Initialize the MathWorks GPU API. */
  mxInitGPU();
  
  if (nrhs != 3)
    mexErrMsgIdAndTxt(errId, errMsg_narg);
  
  
  /* Throw an error if any of the inputs is not a GPU array. */
  if(!mxIsGPUArray(prhs[0]) || !mxIsGPUArray(prhs[1]) || \
          !mxIsGPUArray(prhs[2]))
    mexErrMsgIdAndTxt(errId, errMsg_gpu);

  
  // Get the inputs
  // X 4D input [H W D N] where D is the number of channels and N the 
  // number of images.
  const mxGPUArray *X_mx = mxGPUCreateFromMxArray(prhs[0]);
  // 4D array [H W (D*Nb | Nb) N] or 1D array [D*Nb | Nb]
  const mxGPUArray *Weights_mx  = mxGPUCreateFromMxArray(prhs[1]);
  // 4D array [H W Nb N]
  const mxGPUArray *idx_mx  = mxGPUCreateFromMxArray(prhs[2]);

   
  const mxClassID cid = mxGPUGetClassID(X_mx);
  if (cid != mxGPUGetClassID(Weights_mx))
    mexErrMsgIdAndTxt(errId,errMsg_type);
    
  const mxClassID cid2 = mxGPUGetClassID(idx_mx);
  if (cid2 != mxUINT32_CLASS)
    mexErrMsgIdAndTxt(errId,"The third input must be of type uint32.");
  
  
  // X 4D Array [H W D N]
  const mwSize X_ndims = mxGPUGetNumberOfDimensions(X_mx);
  if ( X_ndims < 1 || X_ndims > 4)
    mexErrMsgIdAndTxt(errId, errMsg_args);
  
  const mwSize *X_dims = mxGPUGetDimensions(X_mx);
  const mwSize H = X_dims[0];
  const mwSize W = (X_ndims < 2) ? 1 : X_dims[1];
  const mwSize D = (X_ndims < 3) ? 1 : X_dims[2];
  const mwSize N = (X_ndims < 4) ? 1 : X_dims[3];
  
  mwSize numElements=H*W*D*N;  
  
  // Idx 4D Array [H W Nb N]
  const mwSize *idx_dims = mxGPUGetDimensions(idx_mx);
  const mwSize idx_ndims = mxGPUGetNumberOfDimensions(idx_mx);
  
  if (idx_ndims > 4)
    mexErrMsgIdAndTxt(errId, errMsg_args);

  if ( N != 1 && idx_ndims != X_ndims)
    mexErrMsgIdAndTxt(errId, errMsg_dims); 
  if ( N == 1 && D != 1 && idx_ndims != X_ndims)
    mexErrMsgIdAndTxt(errId, errMsg_dims); 
  if ( N == 1 && D == 1 && idx_ndims != X_ndims + 1)
    mexErrMsgIdAndTxt(errId, errMsg_dims);   
  
  const mwSize Nb = (idx_ndims < 3) ? 1 : idx_dims[2];
  
  if (idx_dims[0] != X_dims[0])
    mexErrMsgIdAndTxt(errId, errMsg_dims); 
  if (idx_dims[1] != X_dims[1])
    mexErrMsgIdAndTxt(errId, errMsg_dims); 
  if ( N!=1 && idx_dims[3] != X_dims[3])
    mexErrMsgIdAndTxt(errId, errMsg_dims); 
  
    
  // Weights : 4D Array [H W D*Nb N] or [H W Nb N] or 2D Array [D*Nb] or [Nb]
  const mwSize *Weights_dims = mxGPUGetDimensions(Weights_mx);
  const mwSize W_ndims = mxGPUGetNumberOfDimensions(Weights_mx);
  if (W_ndims > 4)
    mexErrMsgIdAndTxt(errId, errMsg_args);
  
  const mwSize numWeightElements = mxGPUGetNumberOfElements(Weights_mx);
  
  bool SpatialSharedWeights = false; 
  bool FMapSharedWeights = false;  
  if (W_ndims == 2){
    SpatialSharedWeights = true; // Weights 2D Array of D*Nb or Nb elements
    
    if (numWeightElements == Nb)
      FMapSharedWeights = true;
    
    if (!FMapSharedWeights && numWeightElements != Nb*D)
      mexErrMsgIdAndTxt(errId, errMsg_dims);    
  }
  
  if (W_ndims > 2){
    if ((Weights_dims[0] != H) || (Weights_dims[1] != W))
      mexErrMsgIdAndTxt(errId, errMsg_dims);
    
    if (Weights_dims[2] == Nb)
      FMapSharedWeights = true;
    
    if (!FMapSharedWeights && Weights_dims[2] != Nb*D)
      mexErrMsgIdAndTxt(errId, errMsg_dims);
    
    if (W_ndims < 4){
      if (N != 1)
        mexErrMsgIdAndTxt(errId, errMsg_dims);
    }
    else{
      if (Weights_dims[3] != N)
        mexErrMsgIdAndTxt(errId, errMsg_dims);
    }
  }
  
  const unsigned int *idx_ptr = static_cast<const unsigned int*>(mxGPUGetDataReadOnly(idx_mx));  
    
  // Create output
  
  // Y [H, W, D, N]
  const mwSize Y_dims[]={H, W, D, N};
  mxGPUArray *Y_mx = mxGPUCreateGPUArray(4, Y_dims, cid, mxREAL, MX_GPU_INITIALIZE_VALUES);
  
  mwSize c_ptr[4]={H*W,H*W*D,D,Nb};
  mwSize *d_c;
  hipMalloc(&d_c,4*sizeof(mwSize));
  hipMemcpy(d_c,c_ptr,4*sizeof(mwSize),hipMemcpyHostToDevice);
  
  
  if (cid == mxDOUBLE_CLASS){
    const double *X_ptr = static_cast<const double*>(mxGPUGetDataReadOnly(X_mx));
    const double *Weights_ptr = static_cast<const double*>(mxGPUGetDataReadOnly(Weights_mx));
    double *Y_ptr = static_cast<double*>(mxGPUGetData(Y_mx));
    
   gpuErrchk(FMapNLSum_gpu<double>(X_ptr,Weights_ptr,Y_ptr,idx_ptr,
               d_c,Nb,FMapSharedWeights,SpatialSharedWeights,numElements));
  }
  else if (cid == mxSINGLE_CLASS){
    const float *X_ptr = static_cast<const float*>(mxGPUGetDataReadOnly(X_mx));
    const float *Weights_ptr = static_cast<const float*>(mxGPUGetDataReadOnly(Weights_mx));
    float *Y_ptr = static_cast<float*>(mxGPUGetData(Y_mx));
    
    gpuErrchk(FMapNLSum_gpu<float>(X_ptr,Weights_ptr,Y_ptr,idx_ptr,
               d_c,Nb,FMapSharedWeights,SpatialSharedWeights,numElements));
    
  }
  else
    mexErrMsgIdAndTxt(errId, errMsg_ty);
  
  
  plhs[0] = mxGPUCreateMxArrayOnGPU(Y_mx);
  
  mxGPUDestroyGPUArray(X_mx);
  mxGPUDestroyGPUArray(Weights_mx);
  mxGPUDestroyGPUArray(Y_mx);
  mxGPUDestroyGPUArray(idx_mx);
  
}










