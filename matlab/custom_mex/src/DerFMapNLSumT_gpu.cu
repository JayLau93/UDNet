#include "hip/hip_runtime.h"
/* // Compilation instructions
 * mex -v DerFMapNLSumT_gpu.cu -largeArrayDims
 * CFLAGS="\$CFLAGS -std=c++11 -g" LDFLAGS='\$LDFLAGS -Wl,-rpath,/usr/local/cuda/lib'
 * -I/usr/local/include/
 * -L/usr/local/cuda/lib -lcudart -lstdc++ -lc */


/* Let us assume that X is of size (H * W) x D x N, idx is of size 
 * (H * W) x Nb x N (computed by misc.patchMatch) where 
 * the elements of idx have values in the range [1, H*W] and Weights is of 
 * size (H x W) x (D x Nb) x N or (H x W) x Nb x N or 
 * D x Nb or Nb. In the second case the same weights are applied on the 
 * D feature maps at each spatial dimension, while in the last two cases  
 * the same weights are applied to the H x W spatial elements.
 *
 * Then the output Y will be of size (H * W) x D x N where 
 *
 *            Nb
 * Y(i,d,n) = Sum W(i,idx_w(d,r),n)*X(idx(i,r,n),d,n), i=1:H*W, d=1:D, n=1:N
 *            r=1
 * and idx_w(d,r) = r*D + d
 *
 *          Nb
 * Y(ind_y) = Sum W(ind_w)*X(ind_x) where
 *          r=1
 * ind_y = i + d*H*W + n*H*W*D, 
 * ind_w = i + (d+r*D)*H*W + n*H*W*D*Nb,  (W is of size HW x D x Nb x N)
 * ind_x = idx[ind_r] + d*H*W + n*H*W*D where 
 * ind_r = i + r*H*W + n*H*W*Nb
 *
 * To check the correctness of the computation do the following in Matlab:
 *
 * H=10;W=20;D=3;N=2;Nb=4;
 * X=randn(H,W,D,N); idx=randsrc(H*W*Nb*N,1,[1:H*W]); idx = reshape(idx,H,W,Nb,N);
 * idx(:,:,1,1) = reshape(1:H*W,H,W); idx(:,:,1,2)=idx(:,:,1,1); idx = uint32(idx);
 * Weights = randn(Nb,1);
 * [X,idx,Weights] = misc.move_data('gpu',X,idx,Weights);
 *
 * Y = zeros(size(X),'like',X);
 * for i = 1:H, for j=1:W, for d=1:D, for n=1:N, for r=1:Nb,
 * Y(i,j,d,n) = Y(i,j,d,n) + Weights(r)*X(idx(i,j,r,n)+(d-1)*H*W+(n-1)*H*W*D);
 * end,end,end,end,end
 *
 * Y2 = WeightedPatchSum_gpu(X,Weights,idx-1); % We substract 1 from idx since
 * % in C the starting element is zero and not 1.
 *
 * e = Y-Y2; max(e(:)), min(e(:))
 *
 * Now in order to compute the adjoint operation let us consider the following
 * example where H*W = 6, D=3, Nb = 2 and N = 1
 *
 * Let X = [ x1:D    idx = [1 3 4   Weights = [ W1:D,1 W1:D,2 W1:D,3
 *           x2:D           2 1 6               W2:D,1 W2:D,2 W2:D,3
 *           x3:D           3 5 2               W3:D,1 W3:D,2 W3:D,3
 *           x4:D           4 1 6               W4:D,1 W4:D,2 W4:D,3
 *           x5:D           5 2 3               W5:D,1 W5:D,2 W5:D,3
 *           x6:D ]         6 4 3]              W6:D,1 W6:D,2 W6:D,3]
 *
 *  xk:D ==> It denotes all the D channels for the k spatial coordinate
 *  W(k,r):D ==> It denotes the D-dimensional weight vector for the rth 
 *  closest neighbor.
 *        
 *  Based on the above, the forward operation is given by:
 *
 *  Y = [ x1:D .* W1:D,1 + x3:D .* W1:D,2 + x4:D .* W1:D,3
 *        x2:D .* W2:D,1 + x1:D .* W2:D,2 + x6:D .* W2:D,3
 *        x3:D .* W3:D,1 + x5:D .* W3:D,2 + x2:D .* W3:D,3
 *        x4:D .* W4:D,1 + x1:D .* W4:D,2 + x6:D .* W4:D,3
 *        x5:D .* W5:D,1 + x2:D .* W5:D,2 + x3:D .* W5:D,3
 *        x6:D .* W6:D,1 + x4:D .* W6:D,2 + x3:D .* W6:D,3 ]
 *
 *  and the adjoint is given by
 *
 *  Z = [ y1:D .* W1:D,1 + y2:D .* W2:D,2 + y4:D .* W4:D,2
 *        y2:D .* W2:D,1 + y3:D .* W3:D,3 + y5:D .* W5:D,2
 *        y1:D .* W1:D,2 + y3:D .* W3:D,1 + y5:D .* W5:D,3 + y6:D .* W6:D,3
 *        y1:D .* W1:D,3 + y4:D .* W4:D,1 + y6:D .* W6:D,2
 *        y2:D .* W2:D,2 + y5:D .* W5:D,1
 *        y2:D .* W2:D,3 + y4:D .* W4:D,3 + y6:D .* W6:D,1 ]
 *
 *  We observe that zio:D,n is given as the weighted sum of Wl:D,m,n and yl:D,n 
 *  where l,m,n are the row, column and slice where the number i+N*n is 
 *  found in idx. (In this example we considered N=1 but this holds true
 *  for N >= 1). 
 *  
 *  NOTE!!!!
 *  In order the adjoint to correctly work for multiple images 
 *  we must transform the idx computed by misc.patchMatch so that 
 *  idx_new(:,:,n) = idx(:,:,n) + (n-1)*H*W. This is taken care by the 
 *  surrogate function FMapNLSumT_helper.m
 *
 *  [widx,n,I]=misc.FMapNLSumT_helper(idx);
 *  Z = FMapNLSumT_gpu(Y,Weights,widx-1,n,I-1);
 *  e = Z(:)'* X(:) - Y(:)'*Y(:) // This should be close to zero.
 *
 *  Now we want to compute the derivative of W w.r.t Z assuming that 
 *  Wi:D,r = W:D,r where r=1:Nb (That is in the spatial dimensions the
 *  weights are shared). This is given by
 *
 *  dZdW = [ y1:D      y2:D y3:D      y4:D y5:D y6:D          [ dZdW:D,1
 *           y2:D+y4:D y5:D y1:D      y6:D y3:D 0           =   dZdW:D,2
 *           0         y3:D y5:D+y6:D y1:D 0    y2:D+y4:D ]     dzdW:D,3 ]
 *           
 * To do that we use the misc.DerFMapNLSumT_helper function to compute
 * the second,third and fourth arguments of the DerWeightedMapSumT function.
 * 
 * Then dW:D,r is computed as follows:
 *  
 * [widx,n,I] = DerFMapNLSumT_helper(idx);
 * dW:D,r = DerFMapNLSumT_gpu(Z,DZDY,widx(:,r,:)-1,n(:,r,:),I(:,r,:)-1);
 * dW:D,r = sum(sum(reshape(dW:D,r,[],D,N),1),3);
 * /
 
/* In a mxArray to access the element X[i][j][z] you can do it by referring
   to the element X[i+j*dims[0]+z*dims[0]*dims[1]] */


#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#if __CUDA_ARCH__ >= 200
#define VL_CUDA_NUM_THREADS 1024
#else
#define VL_CUDA_NUM_THREADS 512
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line)
{
  if (code != hipSuccess)
  {
    char *err_str = new char[1000];
    sprintf(err_str,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    mexErrMsgTxt(err_str);
    delete [] err_str;
  }
}

inline unsigned int divideUpwards(unsigned int a, unsigned int b)
{
  return (a + b - 1) / b ;
}

__device__ void index_map(const mwSize *c, const mwSize index, mwSize *ind){
  
  // From index = i + j*H + d*H*W + n*H*W*D and c = [H*W, H*W*D]
  // we want to recover ind[0] = i + j*H, ind[1]=d and ind[2]=n
 
  ind[0] = index % c[0]; // ind[0] = i + j*H
  ind[1] = ((index - ind[0]) % c[1]) / c[0]; // ind[1] = d
  ind[2] = index / c[1]; // ind[2] = n
         
}

template <typename T>
__global__ void DerFMapNLSumT_gpu_kernel(
        const T *X, const T* Z, T* Y, 
        const unsigned int *n_table, 
        const unsigned int *I, 
        const unsigned int *idx, 
        const mwSize *c,
        size_t numElements)
{
  
  size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  
  if (index < numElements){
    
    mwSize ind[3];
    mwSize i;
    index_map(c,index,ind);
    // For any channel k of X(i,j,k,n) and Y(i,j,k,n) we have to use
    // n_table(i,j,n) and I(i,j,n).
    // index_n = i+j*H+n*H*W
    mwSize index_n = ind[0]+c[0]*ind[2];
    
    for (mwSize k = 0; k < n_table[index_n]; ++k)
    {
      i = idx[I[index_n]+k+c[0]*ind[2]]; // row and column of X
      Y[index] += X[i+c[0]*ind[1]+c[1]*ind[2]];
    }
    Y[index] *= Z[index];
  }
}

template <typename T> 
static inline hipError_t DerFMapNLSumT_gpu(
        const T *X, const T *Z, T *Y, 
        const unsigned int *n_table,
        const unsigned int *I,
        const unsigned int *idx,
        const mwSize *c,
        size_t numElements)
{  
 DerFMapNLSumT_gpu_kernel<T>
 <<< divideUpwards(numElements, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
 (X,Z,Y,n_table,I,idx,c,numElements);
 return hipPeekAtLastError(); 
}

void mexFunction(int nlhs, mxArray *plhs[],
        int nrhs, mxArray const *prhs[])
{
  
  // Create Error Messages.
  char const * const errId = "DerFMapNLSumT_gpu:InvalidInput";
  char const * const errMsg_narg = "Invalid input: Five input "
          "arguments are expected.";
  char const * const errMsg_ty = "Invalid input: Input has unsupported type.";
  char const * const errMsg_args = "Invalid input: All the inputs must be between a 1-D to 4-D array.";
  char const * const errMsg_dims = "Invalid input: Dimensions mismatch.";
  char const * const errMsg_type = "Invalid input: Inputs 1, 2 must be of the same data type.";
  char const * const errMsg_type2 = "Invalid input: Inputs 3-5 must be of type 'uint32'.";
  
  /* Initialize the MathWorks GPU API. */
  mxInitGPU(); 
  
  if (nrhs != 5)
    mexErrMsgIdAndTxt(errId, errMsg_narg);
  
  // Get the inputs
  // X 4D input [H W D N] where D is the number of channels and N the 
  // number of images.
  const mxGPUArray *X_mx = mxGPUCreateFromMxArray(prhs[0]);
  // Z 4D input [H W D N] where D is the number of channels and N the 
  // number of images.  
  const mxGPUArray *Z_mx = mxGPUCreateFromMxArray(prhs[1]);
  // 1D array [H*W*N]
  const mxGPUArray *idx_mx  = mxGPUCreateFromMxArray(prhs[2]);
  // 1D array [H*W*N]
  const mxGPUArray *n_mx  = mxGPUCreateFromMxArray(prhs[3]);
    // 1D array [H*W*N]
  const mxGPUArray *I_mx  = mxGPUCreateFromMxArray(prhs[4]);
  

  const mxClassID cid = mxGPUGetClassID(X_mx);
  if (cid != mxGPUGetClassID(Z_mx))
    mexErrMsgIdAndTxt(errId,errMsg_type);

  const mxClassID cid2 = mxGPUGetClassID(idx_mx);
  if (cid2 != mxUINT32_CLASS)
    mexErrMsgIdAndTxt(errId,errMsg_type2);
  
  if (cid2 != mxGPUGetClassID(n_mx) || cid2 != mxGPUGetClassID(I_mx))
    mexErrMsgIdAndTxt(errId,errMsg_type2);  
  
  // X 4D Array [H W D N]
  const mwSize X_ndims = mxGPUGetNumberOfDimensions(X_mx);
  const mwSize Z_ndims = mxGPUGetNumberOfDimensions(Z_mx);
  if ( X_ndims < 1 || X_ndims > 4)
    mexErrMsgIdAndTxt(errId, errMsg_args);
  
  if ( Z_ndims != X_ndims)
    mexErrMsgIdAndTxt(errId, errMsg_dims);
   
  const mwSize *X_dims = mxGPUGetDimensions(X_mx);
  const mwSize H = X_dims[0];
  const mwSize W = (X_ndims < 2) ? 1 : X_dims[1];
  const mwSize D = (X_ndims < 3) ? 1 : X_dims[2];
  const mwSize N = (X_ndims < 4) ? 1 : X_dims[3];
  
  
  const mwSize *Z_dims = mxGPUGetDimensions(Z_mx);
  for (int k=0; k < X_ndims; ++k){
    if ( Z_dims[k] != X_dims[k])
      mexErrMsgIdAndTxt(errId, errMsg_dims);
  }
  
  mwSize numElements=H*W*D*N;  
   
  // idx Array [H*W*N]
  if (mxGPUGetNumberOfElements(idx_mx)!=H*W*N)
    mexErrMsgIdAndTxt(errId,errMsg_dims);
  const unsigned int *idx_ptr = static_cast<const unsigned int*>(mxGPUGetDataReadOnly(idx_mx));
  
  // n Array [H*W*N]
  if (mxGPUGetNumberOfElements(n_mx)!=H*W*N)
    mexErrMsgIdAndTxt(errId,errMsg_dims);
  const unsigned int *n_ptr = static_cast<const unsigned int*>(mxGPUGetDataReadOnly(n_mx));
  
  // I Array [H*W*N]
  if (mxGPUGetNumberOfElements(I_mx)!=H*W*N)
    mexErrMsgIdAndTxt(errId,errMsg_dims);
  const unsigned int *I_ptr = static_cast<const unsigned int*>(mxGPUGetDataReadOnly(I_mx));

    
  // Create output
  
  // Y [H, W, D, N]
  const mwSize Y_dims[]={H, W, D, N};
  mxGPUArray *Y_mx = mxGPUCreateGPUArray(4, Y_dims, cid, mxREAL, MX_GPU_INITIALIZE_VALUES);
  
  const mwSize c_ptr[2]={H*W,H*W*D};
  mwSize *d_c;
  hipMalloc(&d_c,2*sizeof(mwSize));
  hipMemcpy(d_c,c_ptr,2*sizeof(mwSize),hipMemcpyHostToDevice);

  
  if (cid == mxDOUBLE_CLASS){
    const double *X_ptr = static_cast<const double*>(mxGPUGetDataReadOnly(X_mx));
    const double *Z_ptr = static_cast<const double*>(mxGPUGetDataReadOnly(Z_mx));
    double *Y_ptr = static_cast<double*>(mxGPUGetData(Y_mx));
    
    gpuErrchk(DerFMapNLSumT_gpu<double>(X_ptr,Z_ptr,Y_ptr,n_ptr,I_ptr,idx_ptr,d_c,numElements));
  }
  else if (cid == mxSINGLE_CLASS){
    const float *X_ptr = static_cast<const float*>(mxGPUGetDataReadOnly(X_mx));
    const float *Z_ptr = static_cast<const float*>(mxGPUGetDataReadOnly(Z_mx));
    float *Y_ptr = static_cast<float*>(mxGPUGetData(Y_mx));
    
    gpuErrchk(DerFMapNLSumT_gpu<float>(X_ptr,Z_ptr,Y_ptr,n_ptr,I_ptr,idx_ptr,d_c,numElements));
    
  }
  else
    mexErrMsgIdAndTxt(errId, errMsg_ty);
  
  
  plhs[0] = mxGPUCreateMxArrayOnGPU(Y_mx);
  
  mxGPUDestroyGPUArray(X_mx);
  mxGPUDestroyGPUArray(Z_mx);
  mxGPUDestroyGPUArray(Y_mx);
  mxGPUDestroyGPUArray(idx_mx);
  mxGPUDestroyGPUArray(n_mx);
  mxGPUDestroyGPUArray(I_mx);
}










